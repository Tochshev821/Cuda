#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <math.h>
#include <vector>



__global__ void MonteKarloPi(int * niter, int *count, double* x, double* y)
{

    for (int i = 0; i < *niter; ++i)
    {   

        //get random points

        
        double z = (x[i]* x[i])+ (y[i]* y[i]);

        //check to see if point is in unit circle
        if (z <= 1)
        {
            ++*count;
        }
    }
    
    
}



int main()
{


    const int niter = 10000;
    int i;
    int count = 0;
    double pi;
    int* dev_niter, *dev_count;
    int size = sizeof(int);
    double x[niter], *dev_x;
    double y[niter], *dev_y;
    const size_t x_size = sizeof(double) * size_t(niter);

    srand(time(NULL));

    for (int i = 0; i < niter; i++)
    {
        x[i] = (double)rand() / RAND_MAX;
        y[i] = (double)rand() / RAND_MAX;

    }
    
    hipMalloc((void**)&dev_niter, size);
    hipMalloc((void**)&dev_count, size);

    hipMalloc((void**)&dev_x, x_size);
    hipMalloc((void**)&dev_y, x_size);

    hipMemcpy(dev_niter, &niter, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_count, &count, size, hipMemcpyHostToDevice);

    hipMemcpy(dev_x, x, x_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, x_size, hipMemcpyHostToDevice);

    MonteKarloPi << < 1, 1 >> > (dev_niter, dev_count, dev_x, dev_y);
    
    hipMemcpy( &count, dev_count, size, hipMemcpyDeviceToHost);



    pi = ((double)count / (double)niter) * 4.0;
    printf("Pi: %f\n", pi);


	return 0;
}




